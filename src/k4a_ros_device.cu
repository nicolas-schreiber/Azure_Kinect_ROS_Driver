#include "hip/hip_runtime.h"
#include "azure_kinect_ros_driver/k4a_ros_device.h"

#include <thrust/device_vector.h>

struct __attribute__((__packed__)) i16Point {
    int16_t x;
    int16_t y;
    int16_t z;
};

struct __attribute__((__packed__)) ui8RGBA {
    uint8_t b;
    uint8_t g;
    uint8_t r;
    uint8_t a;
};

/**
 * Point Struct
 * Represents required structure from PointXYZRGB PointCloud ROS Messages
 * Including Padding of the two parts rgb and xyz into two WORDS
 */
struct PointXYZRGB {
  float x = 0;
  float y = 0;
  float z = 0;
  float _pad0;
  float rgb = 0;
  float _pad_1;
  float _pad_2;
  float _pad_3;

  /// Empty Constructor for initial declarations
  __host__ __device__
  inline PointXYZRGB(){};

  /// Constructor from values
  __host__ __device__
  inline PointXYZRGB(float x_, float y_, float z_, float rgb_) 
    : x(x_), y(y_), z(z_), rgb(rgb_) {};

  /// Constructor from individual rgb
  __host__ __device__
  inline PointXYZRGB(float x_, float y_, float z_, uint8_t r, uint8_t g, uint8_t b) : x(x_), y(y_), z(z_)
  {
    uint8_t* rgb_ = (uint8_t*) &rgb;
    rgb_[0] = r;
    rgb_[1] = g;
    rgb_[2] = b;
    rgb_[3] = 0;
  };
};

/**
 * Point Struct
 * Represents required structure from PointXYZ PointCloud ROS Messages
 * Including Padding of the xyz values
 */
struct PointXYZ {
  float x = 0;
  float y = 0;
  float z = 0;
  float _pad;

  /// Empty Constructor for initial declarations
  __host__ __device__
  inline PointXYZ(){};

  /// Constructor from values
  __host__ __device__
  inline PointXYZ(float x_, float y_, float z_) : x(x_), y(y_), z(z_) {};
};



struct RGBDToPointXYZRGB {
  __host__ __device__ 
  PointXYZRGB operator()(i16Point point, ui8RGBA color) {
    // Check Point Validity
    if (point.z <= 0.0f || color.a == 0)
      return PointXYZRGB(nanf(""), nanf(""), nanf(""), 0);
    
    // Create Point
    constexpr float kMillimeterToMeter = 1.0 / 1000.0f;
    float x = kMillimeterToMeter * static_cast<float>(point.x);
    float y = kMillimeterToMeter * static_cast<float>(point.y);
    float z = kMillimeterToMeter * static_cast<float>(point.z);

    return PointXYZRGB(x, y, z, color.r, color.g, color.b);
  }
};

struct DToPointXYZ {
  __host__ __device__ 
  PointXYZ operator()(i16Point point) {
    // Check Point Validity
    if (point.z <= 0.0f)
      return PointXYZ(nanf(""), nanf(""), nanf(""));
    
    // Create Point
    constexpr float kMillimeterToMeter = 1.0 / 1000.0f;
    float x = kMillimeterToMeter * static_cast<float>(point.x);
    float y = kMillimeterToMeter * static_cast<float>(point.y);
    float z = kMillimeterToMeter * static_cast<float>(point.z);

    return PointXYZ(x, y, z);
  }
};


void cudaFillColorPointCloud(const k4a::image& pointcloud_image, const k4a::image& color_image, sensor_msgs::PointCloud2Ptr& point_cloud) 
{
  const size_t point_count = pointcloud_image.get_height_pixels() * pointcloud_image.get_width_pixels();

  const i16Point* h_points = reinterpret_cast<const i16Point*>(pointcloud_image.get_buffer());
  const ui8RGBA*  h_colors = reinterpret_cast<const ui8RGBA*> (color_image.get_buffer());
  PointXYZRGB* h_out = reinterpret_cast<PointXYZRGB*>(point_cloud->data.data());

  thrust::device_vector<i16Point> d_points(point_count);
  thrust::device_vector<ui8RGBA>  d_colors(point_count);
  thrust::device_vector<PointXYZRGB> d_out(point_count);

  thrust::copy(h_points, h_points + point_count, d_points.begin());
  thrust::copy(h_colors, h_colors + point_count, d_colors.begin());

  thrust::transform(d_points.begin(), d_points.end(), d_colors.begin(), d_out.begin(), RGBDToPointXYZRGB());

  thrust::copy(d_out.begin(), d_out.end(), h_out); 
}

void cudaFillPointCloud(const k4a::image& pointcloud_image, sensor_msgs::PointCloud2Ptr& point_cloud) 
{
  const size_t point_count = pointcloud_image.get_height_pixels() * pointcloud_image.get_width_pixels();

  const i16Point* h_points = reinterpret_cast<const i16Point*>(pointcloud_image.get_buffer());
  PointXYZ* h_out = reinterpret_cast<PointXYZ*>(point_cloud->data.data());

  thrust::device_vector<i16Point> d_points(point_count);
  thrust::device_vector<PointXYZ> d_out(point_count);

  thrust::copy(h_points, h_points + point_count, d_points.begin());

  thrust::transform(d_points.begin(), d_points.end(), d_out.begin(), DToPointXYZ());

  thrust::copy(d_out.begin(), d_out.end(), h_out); 
}



